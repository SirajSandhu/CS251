
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define CUDA_ERROR_EXIT(str)  do{\
                                hipError_t err = hipGetLastError();\
                                if( err != hipSuccess){\
                                  printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                  exit(-1);\
                                }\
                              }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

__global__ void compute(int *mem, int n, int skip) {
    int index;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i >= n) return;
    index = i/skip;
    if(i % skip == 0) {
        if(index % 2) { //odd index
            //store at the end;
            if(i + skip - 1 >= n) {
                if(i != n - 1) {
                    *(mem + (n - 1)) =  *(mem + (n - 1)) ^ *(mem + (i));
                    *(mem + i) = 0; //because a ^ 0 = a;
                }
            }
            else {
                *(mem + (i + skip - 1)) =  *(mem + (i + skip - 1)) ^ *(mem + (i));
                *(mem + i) = 0;     //because a ^ 0 = a;
            }
        }
        else {
            if(i + skip - 1 >= n) {
                if(i != n - 1) {
                    *(mem + i) =  *(mem + (n - 1)) ^ *(mem + (i));
                    *(mem + (n - 1)) = 0; //because a ^ 0 = a;
                }
            }
            else {
                *(mem + i) =  *(mem + (i)) ^ *(mem + (i + skip - 1));
                *(mem + (i + skip - 1)) = 0; //because a ^ 0 = a;
            }
        }
    }
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int ctr;
    int *a;
    int *gpu_mem;
    int blocks;

    int n = atoi(argv[1]);
    int seed = atoi(argv[2]);

    /* Allocate host (CPU) memory and initialize*/
    a = (int*)malloc(n * sizeof(int));

    srand(seed);
    for (ctr = 0; ctr < n; ++ctr)
        a[ctr] = random();

    gettimeofday(&t_start, NULL);

    /* Allocate GPU memory and copy from CPU --> GPU*/
    hipMalloc(&gpu_mem, n * sizeof(int));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, a, n * sizeof(int) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");

    gettimeofday(&start, NULL);

    int skip;
    blocks = n /1024;
    if (n % 1024)
        ++blocks;
    for (skip = 2; skip < 2*n; skip *= 2)
        compute<<<blocks, 1024>>>(gpu_mem, n, skip);

    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);

    /* Copy back result*/
    hipMemcpy(a, gpu_mem, n * sizeof(int) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");

    gettimeofday(&t_end, NULL);

    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);

    /*int *number = (int *) (a);*/
    printf("%d \n", *(a));

    free(a);
}
